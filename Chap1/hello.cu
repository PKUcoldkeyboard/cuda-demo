#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "dbg.h"

__global__ void helloFromGPU(void)
{
    int threadId = threadIdx.x;
    printf("Hello from GPU! threadId: %d\n", threadId);
}

int main(int argc, char **argv)
{
    hipError_t cudaStatus = hipSuccess;
    printf("Hello from CPU!\n");

    int blockDim = 10;
    int gridDim  = 1;
    helloFromGPU<<<gridDim, blockDim>>>();
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        log_err("Failed to synchronize, %s\n", hipGetErrorString(cudaStatus));
        hipDeviceReset();
        return -1;
    }
    hipDeviceReset();
    return 0;
}