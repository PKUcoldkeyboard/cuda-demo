#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <time.h>
#include "dbg.h"

hipEvent_t start, stop;
clock_t hstart, hstop;

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1e-8;
    bool match     = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            log_err("Arrays do not match!");
            log_err("host %5.2f gpu %5.2f at current %d", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match)
    {
        log_info("Arrays match.");
    }
}

void init(float *A, float *B, const int N)
{
    for (int i = 0; i < N; i++)
    {
        A[i] = ((float)i) + 0.1335f;
        B[i] = 1.50f * ((float)i) + 0.9383f;
    }
}

__global__ void sumArraysOnGPUKernel(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int size = N * sizeof(float);
    float *d_A, *d_B, *d_C;

    check_device(hipMalloc((void **)&d_A, size));
    check_device(hipMalloc((void **)&d_B, size));
    check_device(hipMalloc((void **)&d_C, size));

    check_device(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    check_device(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

    int threadsPerBlock = 512;
    int blocksPerGrid   = (N + threadsPerBlock - 1) / threadsPerBlock;

    check_device(hipEventCreate(&start));
    check_device(hipEventCreate(&stop));
    check_device(hipEventRecord(start, 0));
    sumArraysOnGPUKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    check_device(hipEventRecord(stop, 0));
    check_device(hipEventSynchronize(stop));
    check_device(hipDeviceSynchronize());

    check_device(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));

    check_device(hipFree(d_A));
    check_device(hipFree(d_B));
    check_device(hipFree(d_C));
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    hstart = clock();
    for (int i = 0; i < N; i++)
    {
        C[i] = A[i] + B[i];
    }
    hstop = clock();
}

int main(int argc, char **argv)
{
    log_info("[CUDA] sumArraysOnGPU...");
    int dev = 0;
    hipDeviceProp_t deviceProp;
    check_device(hipGetDeviceProperties(&deviceProp, dev));
    log_info("Using Device %d: %s", dev, deviceProp.name);
    check_device(hipSetDevice(dev));

    int nElem = 1 << 24;
    log_info("Vector size %d", nElem);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nElem * sizeof(float));
    h_B     = (float *)malloc(nElem * sizeof(float));
    hostRef = (float *)malloc(nElem * sizeof(float));
    gpuRef  = (float *)malloc(nElem * sizeof(float));

    init(h_A, h_B, nElem);
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    sumArraysOnGPU(h_A, h_B, gpuRef, nElem);

    checkResult(hostRef, gpuRef, nElem);

    float gpuTime = 0.0f;
    check_device(hipEventElapsedTime(&gpuTime, start, stop));
    log_info("sumArraysOnGPU elapsed %f ms", gpuTime);

    log_info("sumArraysOnHost time elapsed %f ms",
             (double)(hstop - hstart) / CLOCKS_PER_SEC * 1000.0);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    return 0;
}