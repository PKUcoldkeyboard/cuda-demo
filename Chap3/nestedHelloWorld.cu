#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "dbg.h"

/**
* This example demonstrates the use of recursion in CUDA
* The kernel launches itself recursively until the value of iSize is 1
*/
__global__ void nestedHelloWorld(const int iSize, int iDepth) {
    int tid = threadIdx.x;
    printf("Recursion=%d: Hello World from thread %d block %d\n", iDepth, tid, blockIdx.x);

    if (iSize == 1) {
        return;
    }

    int nThreads = iSize >> 1;

    if (tid == 0 && nThreads > 0) {
        nestedHelloWorld<<<1, nThreads>>>(nThreads, ++iDepth);
        printf("------> nested execution depth: %d\n", iDepth);
    }
}

int main(int argc, char **argv) {
    int size = 8;
    int blockSize = 8;
    int igrid = 1;

    if (argc > 1) {
        igrid = atoi(argv[1]);
        size = igrid * blockSize;
    }

    dim3 block(blockSize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    log_info("%s Execution Configuration: grid %d block %d", argv[0], grid.x, block.x);

    nestedHelloWorld<<<grid, block>>>(block.x, 0);

    CHECK(hipGetLastError());
    CHECK(hipDeviceReset());
    return 0;
}