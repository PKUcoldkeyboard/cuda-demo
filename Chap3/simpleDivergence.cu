#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "dbg.h"

/**
* branch efficiency： 83.33%
*/
__global__ void mathKernel1(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if (tid % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

/**
* best branch efficiency： 100%
*/
__global__ void mathKernel2(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

/**
*  branch efficiency： 71.43%
*/
__global__ void mathKernel3(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    bool ipred = (tid % 2 == 0);

    if (ipred)
    {
        ia = 100.0f;
    }

    if (!ipred)
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel4(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    int itid = tid >> 5;

    if (itid & 0x01 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void warmingup(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

int main(int argc, char **argv) {
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    log_info("%s using Device %d: %s", argv[0], dev, deviceProp.name);

    int size = 64;
    int blockSize = 64;

    if (argc > 1) {
        blockSize = atoi(argv[1]);
    }

    if (argc > 2) {
        size = atoi(argv[2]);
    }

    log_info("Data size %d ", size);

    dim3 block(blockSize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    log_info("Execution Configure (block %d grid %d)", block.x, grid.x);

    float *d_C;
    size_t nBytes = size * sizeof(float);
    CHECK(hipMalloc((float**)&d_C, nBytes));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start, 0));
    warmingup<<<grid, block>>>(d_C);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float elapsedTime;
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    log_info("warmup <<<%d, %d>>> elapsed %f ms", grid.x, block.x, elapsedTime);

    CHECK(hipGetLastError());
    CHECK(hipEventRecord(start, 0));
    mathKernel1<<<grid, block>>>(d_C);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    log_info("mathKernel <<<%d, %d>>> elapsed %f ms", grid.x, block.x, elapsedTime);
    CHECK(hipGetLastError());

    CHECK(hipEventRecord(start, 0));
    mathKernel2<<<grid, block>>>(d_C);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    log_info("mathKernel2 <<<%d, %d>>> elapsed %f ms", grid.x, block.x, elapsedTime);
    CHECK(hipGetLastError());

    CHECK(hipEventRecord(start, 0));
    mathKernel3<<<grid, block>>>(d_C);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    log_info("mathKernel3 <<<%d, %d>>> elapsed %f ms", grid.x, block.x, elapsedTime);
    CHECK(hipGetLastError());

    CHECK(hipEventRecord(start, 0));
    mathKernel4<<<grid, block>>>(d_C);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    log_info("mathKernel4 <<<%d, %d>>> elapsed %f ms", grid.x, block.x, elapsedTime);
    CHECK(hipGetLastError());

    CHECK(hipFree(d_C));
    CHECK(hipDeviceReset());
    return 0;
}